#include <hip/hip_runtime.h>

#include <iostream>
#include <random>
using namespace std;

void matmul(const float *a, int ah, int aw, const float *b, int bw, float *c, float alpha, float beta) {
    for (int i = 0; i < ah; i++) {
        for (int j = 0; j < bw; j++) {
            float sum = 0;
            for (int k = 0; k < aw; k++)
                sum += a[i * aw + k] * b[k * bw + j];
            c[i * bw + j] = alpha * sum + beta * c[i * bw + j];
        }
    }
}

__global__ void matmul_kernel(
    const float *A, int Ah, int Aw,
    const float *B, int Bw,
    float *C,
    float alpha, float beta) {
    const int BM = 64;
    const int BK = 16;
    auto tx = threadIdx.x % 32;
    auto ty = threadIdx.x / 32;
    // auto x = blockIdx.x * BLOCKSIZE + tx;
    // auto y = blockIdx.y * BLOCKSIZE + ty;

    auto block_y_begin = (blockIdx.y * BM)*Aw;
    auto block_y_end = block_y_begin + Aw;
    auto block_y_step = BK;
    auto block_x_begin = blockIdx.x * BM;
    auto block_x_step = BK * Bw;

    auto innerAy = threadIdx.x / BK;
    auto innerAx = threadIdx.x % BK;
    auto innerBy = threadIdx.x / BM;
    auto innerBx = threadIdx.x % BM;

    float tmp[4] = {0.0};
    for (int a_bg = block_y_begin, b_bg = block_x_begin; a_bg < block_y_end; a_bg += block_y_step, b_bg += block_x_step) {
        __shared__ float As[BM * BK];
        __shared__ float Bs[BK * BM];
        As[innerAy * BK + innerAx] = A[a_bg + innerAy * Aw + innerAx];
        Bs[innerBy * BM + innerBx] = B[b_bg + innerBy * Bw + innerBx];
        __syncthreads();
#pragma unroll
        for (int m=0; m<2; m++) {
#pragma unroll
            for(int n=0;n<2;n++) {
                auto As_ = As + m * 32 * BK;
                auto Bs_ = Bs + n * 32;
                for (int k = 0; k < BK; ++k) {
                    tmp[m*2+n] += As_[ty * BK + k] * Bs_[k * BM + tx];
                }
            }
        }    
        __syncthreads();
    }
#pragma unroll
    for (int m=0; m<2; m++) {
#pragma unroll
        for(int n=0;n<2;n++) {
            auto x = blockIdx.x * 64 + n * 32 + tx;
            auto y = blockIdx.y * 64 + m * 32 + ty;
            C[y * Bw + x] = alpha * tmp[m*2+n] + beta * C[y * Bw + x];
        }
    }
}

float matmul_cu(const float *a, int ah, int aw, const float *b, int bw, float *c, float alpha, float beta) {
    dim3 threadsPerBlock(32 * 32);
    dim3 numBlocks(ah / 64, bw / 64);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matmul_kernel<<<numBlocks, threadsPerBlock>>>(a, ah, aw, b, bw, c, alpha, beta);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << milliseconds << " ms" << std::endl;
    return milliseconds;
}

int main() {
    const int ah = 1024;
    const int aw = 1024;
    const int bw = 1024;
    const float alpha = 0.5;
    const float beta = 0.5;

    auto ref_a = new float[ah * aw];
    auto ref_b = new float[aw * bw];
    auto ref_c = new float[ah * bw];
    for (int i = 0; i < ah * aw; i++)
        ref_a[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    for (int i = 0; i < aw * bw; i++)
        ref_b[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    for (int i = 0; i < ah * bw; i++)
        ref_c[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);

    float *a, *b, *c;
    hipMalloc(&a, ah * aw * sizeof(float));
    hipMalloc(&b, aw * bw * sizeof(float));
    hipMalloc(&c, ah * bw * sizeof(float));
    hipMemcpy(a, ref_a, ah * aw * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b, ref_b, aw * bw * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c, ref_c, ah * bw * sizeof(float), hipMemcpyHostToDevice);

    matmul(ref_a, ah, aw, ref_b, bw, ref_c, alpha, beta);
    auto timems = matmul_cu(a, ah, aw, b, bw, c, alpha, beta);
    float total_GBytes = (ah * aw + aw * bw + ah * bw + ah * bw) * sizeof(float) / 1024.0 / 1024 / 1024;
    std::cout << total_GBytes / (timems/1000.0) << " gbps\n";

    auto out_c = new float[ah * bw];
    hipMemcpy(out_c, c, ah * bw * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < ah * bw; i++) {
        auto diff = out_c[i] - ref_c[i];
        diff = diff > 0 ? diff : -diff;
        // std::cout<<diff<<"\n";
        if (diff > 0.1)
            return 1;
    }
    std::cout << "ok\n";

    hipFree(a);
    hipFree(b);
    hipFree(c);
    delete ref_a;
    delete ref_b;
    delete ref_c;
    delete out_c;
    return 0;
}
