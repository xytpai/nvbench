#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <random>
#include <vector>
#include <assert.h>
#include "cutlass/gemm/device/gemm.h"

template <typename scalar_t>
float gemm_cuda(
    scalar_t *out,
    const scalar_t *a,
    const scalar_t *b,
    const int m, const int n, const int k,
    const scalar_t alpha,
    const scalar_t beta) {
    using RowMajor = cutlass::layout::RowMajor;
    using CutlassGemm = cutlass::gemm::device::Gemm<float, RowMajor, float, RowMajor, float, RowMajor>;
    CutlassGemm gemm_op;
    CutlassGemm::Arguments args(
        {m, n, k},
        {a, k},
        {b, n},
        {out, n},
        {out, n},
        {alpha, beta}
    );
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    gemm_op(args);
    hipDeviceSynchronize();
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    return milliseconds;
}

template <typename scalar_t = float>
__global__ void gemm_cuda_ref_kernel(
    scalar_t *out,
    const scalar_t *a,
    const scalar_t *b,
    const int m, const int n, const int k,
    const scalar_t alpha,
    const scalar_t beta) {
    auto mi = blockIdx.y * 32 + threadIdx.y;
    auto ni = blockIdx.x * 32 + threadIdx.x;
    if (mi < m && ni < n) {
        float acc = 0.f;
        for (int ki = 0; ki < k; ki++) {
            acc += a[mi * k + ki] * b[ki * n + ni];
        }
        out[mi * n + ni] = alpha * acc + beta * out[mi * n + ni];
    }
}

template <typename scalar_t = float>
void gemm_cuda_ref(
    scalar_t *out,
    const scalar_t *a,
    const scalar_t *b,
    const int m, const int n, const int k,
    const scalar_t alpha,
    const scalar_t beta) {
    dim3 block(32, 32);
    dim3 grid((n + 32 - 1) / 32, (m + 32 - 1) / 32);
    gemm_cuda_ref_kernel<scalar_t><<<grid, block>>>(out, a, b, m, n, k, alpha, beta);
    hipDeviceSynchronize();
}

struct gemm_sizes {
    int m, n, k;
    float alpha, beta;
    gemm_sizes(int m_, int n_, int k_, float a, float b) :
        m(m_), n(n_), k(k_), alpha(a), beta(b) {
    }
};

int main() {
    std::cout << "sgemm_cutlass\n";
    using scalar_t = float;

    std::vector<gemm_sizes> sizes;
    sizes.push_back(gemm_sizes(512, 512, 512, 0.5, 0.5));
    sizes.push_back(gemm_sizes(1024, 1024, 1024, 0.5, 0.5));
    sizes.push_back(gemm_sizes(1028, 1028, 1028, 0.5, 0.5));
    sizes.push_back(gemm_sizes(2048, 2048, 2048, 0.5, 0.5));
    sizes.push_back(gemm_sizes(4096, 4096, 4096, 0.5, 0.5));
    sizes.push_back(gemm_sizes(8192, 8192, 8192, 0.5, 0.5));
    sizes.push_back(gemm_sizes(1<<14, 1<<14, 1<<14, 0.5, 0.5));

    for (auto size : sizes) {
        int m = size.m;
        int n = size.n;
        int k = size.k;
        auto alpha = size.alpha;
        auto beta = size.beta;

        std::cout << "m=" << m << ", n=" << n << ", k=" << k
                  << ", alpha=" << alpha << ", beta=" << beta << "\n";

        auto a_cpu = new scalar_t[m * k];
        auto b_cpu = new scalar_t[k * n];
        auto out_cpu = new scalar_t[m * n];
        for (int i = 0; i < m * k; i++)
            a_cpu[i] = static_cast<scalar_t>(rand()) / static_cast<scalar_t>(RAND_MAX);
        for (int i = 0; i < k * n; i++)
            b_cpu[i] = static_cast<scalar_t>(rand()) / static_cast<scalar_t>(RAND_MAX);
        for (int i = 0; i < m * n; i++)
            out_cpu[i] = static_cast<scalar_t>(rand()) / static_cast<scalar_t>(RAND_MAX);

        scalar_t *a_cuda, *b_cuda, *out_cuda;
        hipMalloc(&a_cuda, m * k * sizeof(scalar_t));
        hipMalloc(&b_cuda, k * n * sizeof(scalar_t));
        hipMalloc(&out_cuda, m * n * sizeof(scalar_t));
        hipMemcpy(a_cuda, a_cpu, m * k * sizeof(scalar_t), hipMemcpyHostToDevice);
        hipMemcpy(b_cuda, b_cpu, k * n * sizeof(scalar_t), hipMemcpyHostToDevice);
        hipMemcpy(out_cuda, out_cpu, m * n * sizeof(scalar_t), hipMemcpyHostToDevice);

        scalar_t *a_cuda_ref, *b_cuda_ref, *out_cuda_ref;
        hipMalloc(&a_cuda_ref, m * k * sizeof(scalar_t));
        hipMalloc(&b_cuda_ref, k * n * sizeof(scalar_t));
        hipMalloc(&out_cuda_ref, m * n * sizeof(scalar_t));
        hipMemcpy(a_cuda_ref, a_cpu, m * k * sizeof(scalar_t), hipMemcpyHostToDevice);
        hipMemcpy(b_cuda_ref, b_cpu, k * n * sizeof(scalar_t), hipMemcpyHostToDevice);
        hipMemcpy(out_cuda_ref, out_cpu, m * n * sizeof(scalar_t), hipMemcpyHostToDevice);

        gemm_cuda_ref<scalar_t>(out_cuda_ref, a_cuda_ref, b_cuda_ref, m, n, k, alpha, beta);
        auto timems = gemm_cuda<scalar_t>(out_cuda, a_cuda, b_cuda, m, n, k, alpha, beta);

        double total_gbytes = ((double)m * k + k * n + m * n + m * n) * sizeof(scalar_t) / 1000.0 / 1000 / 1000;
        std::cout << timems << " ms, " << total_gbytes / (timems / 1000.0) << " gbps, ";

        double tflops = ((double)2 * m * n * k) / (timems / 1000) * 1e-12;
        std::cout << tflops << " tflops\n";

        auto out_cuda_ref_ = new scalar_t[m * n];
        auto out_cuda_ = new scalar_t[m * n];
        hipMemcpy(out_cuda_ref_, out_cuda_ref, m * n * sizeof(scalar_t), hipMemcpyDeviceToHost);
        hipMemcpy(out_cuda_, out_cuda, m * n * sizeof(scalar_t), hipMemcpyDeviceToHost);
        auto maxdiff = -std::numeric_limits<scalar_t>::infinity();
        for (int i = 0; i < m * n; i++) {
            // if (i < 100)
            //     std::cout << out_cuda_[i] << " " << out_cuda_ref_[i] << "\n";
            auto diff = std::abs(out_cuda_[i] - out_cuda_ref_[i]);
            maxdiff = std::max(maxdiff, diff);
        }
        std::cout << "maxdiff: " << maxdiff << std::endl;

        hipFree(a_cuda);
        hipFree(b_cuda);
        hipFree(out_cuda);
        hipFree(a_cuda_ref);
        hipFree(b_cuda_ref);
        hipFree(out_cuda_ref);
        delete[] a_cpu;
        delete[] b_cpu;
        delete[] out_cpu;
        delete[] out_cuda_;
    }
    return 0;
}
