#include <hip/hip_runtime.h>

#include <iostream>
#include <random>
using namespace std;

void matmul(const float *a, int ah, int aw, const float *b, int bw, float *c, float alpha, float beta) {
    for (int i = 0; i < ah; i++) {
        for (int j = 0; j < bw; j++) {
            float sum = 0;
            for (int k = 0; k < aw; k++)
                sum += a[i * aw + k] * b[k * bw + j];
            c[i * bw + j] = alpha * sum + beta * c[i * bw + j];
        }
    }
}

__global__ void matmul_kernel(
    const float *A, int Ah, int Aw,
    const float *B, int Bw,
    float *C,
    float alpha, float beta) {
    const int BLOCKSIZE = 32;
    auto tx = threadIdx.x % BLOCKSIZE;
    auto ty = threadIdx.x / BLOCKSIZE;
    auto x = blockIdx.x * BLOCKSIZE + tx;
    auto y = blockIdx.y * BLOCKSIZE + ty;

    auto block_y_begin = (blockIdx.y * BLOCKSIZE)*Aw;
    auto block_y_end = block_y_begin + Aw;
    auto block_y_step = BLOCKSIZE;
    auto block_x_begin = blockIdx.x * BLOCKSIZE;
    auto block_x_step = BLOCKSIZE * Bw;

    float tmp = 0.0;
    for (int a_bg = block_y_begin, b_bg = block_x_begin; a_bg < block_y_end; a_bg += block_y_step, b_bg += block_x_step) {
        __shared__ float As[32 * 32];
        __shared__ float Bs[32 * 32];
        As[threadIdx.x] = A[a_bg + ty * Aw + tx];
        Bs[threadIdx.x] = B[b_bg + ty * Bw + tx];
        __syncthreads();
        for (int k = 0; k < BLOCKSIZE; ++k)
            tmp += As[ty * BLOCKSIZE + k] * Bs[k * BLOCKSIZE + tx];
        __syncthreads();
    }
    C[y * Bw + x] = alpha * tmp + beta * C[y * Bw + x];
}

float matmul_cu(const float *a, int ah, int aw, const float *b, int bw, float *c, float alpha, float beta) {
    dim3 threadsPerBlock(32 * 32);
    dim3 numBlocks(ah / 32, bw / 32);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matmul_kernel<<<numBlocks, threadsPerBlock>>>(a, ah, aw, b, bw, c, alpha, beta);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << milliseconds << " ms" << std::endl;
    return milliseconds;
}

int main() {
    const int ah = 1024;
    const int aw = 1024;
    const int bw = 1024;
    const float alpha = 0.5;
    const float beta = 0.5;

    auto ref_a = new float[ah * aw];
    auto ref_b = new float[aw * bw];
    auto ref_c = new float[ah * bw];
    for (int i = 0; i < ah * aw; i++)
        ref_a[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    for (int i = 0; i < aw * bw; i++)
        ref_b[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    for (int i = 0; i < ah * bw; i++)
        ref_c[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);

    float *a, *b, *c;
    hipMalloc(&a, ah * aw * sizeof(float));
    hipMalloc(&b, aw * bw * sizeof(float));
    hipMalloc(&c, ah * bw * sizeof(float));
    hipMemcpy(a, ref_a, ah * aw * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b, ref_b, aw * bw * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c, ref_c, ah * bw * sizeof(float), hipMemcpyHostToDevice);

    matmul(ref_a, ah, aw, ref_b, bw, ref_c, alpha, beta);
    auto timems = matmul_cu(a, ah, aw, b, bw, c, alpha, beta);
    float total_GBytes = (ah * aw + aw * bw + ah * bw + ah * bw) * sizeof(float) / 1024.0 / 1024 / 1024;
    std::cout << total_GBytes / (timems/1000.0) << " gbps\n";

    auto out_c = new float[ah * bw];
    hipMemcpy(out_c, c, ah * bw * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < ah * bw; i++) {
        auto diff = out_c[i] - ref_c[i];
        diff = diff > 0 ? diff : -diff;
        // std::cout<<diff<<"\n";
        if (diff > 0.1)
            return 1;
    }
    std::cout << "ok\n";

    hipFree(a);
    hipFree(b);
    hipFree(c);
    delete ref_a;
    delete ref_b;
    delete ref_c;
    delete out_c;
    return 0;
}