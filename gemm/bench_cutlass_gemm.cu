#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <random>
#include "cutlass/gemm/device/gemm.h"
using namespace std;

void matmul(const float *a, int ah, int aw, const float *b, int bw, float *c, float alpha, float beta) {
    for (int i = 0; i < ah; i++) {
        for (int j = 0; j < bw; j++) {
            float sum = 0;
            for (int k = 0; k < aw; k++)
                sum += a[i * aw + k] * b[k * bw + j];
            c[i * bw + j] = alpha * sum + beta * c[i * bw + j];
        }
    }
}

float matmul_cu(const float *a, int ah, int aw, const float *b, int bw, float *c, float alpha, float beta) {
    using RowMajor = cutlass::layout::RowMajor;
    using CutlassGemm = cutlass::gemm::device::Gemm<float, RowMajor, float, RowMajor, float, RowMajor>;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    CutlassGemm gemm_op;
    CutlassGemm::Arguments args(
        {ah, bw, aw},
        {a, aw},
        {b, bw},
        {c, bw},
        {c, bw},
        {alpha, beta}
    );
    gemm_op(args);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << milliseconds << " ms" << std::endl;
    return milliseconds;
}

int main() {
    const int ah = 1024;
    const int aw = 1024;
    const int bw = 1024;
    const float alpha = 0.5;
    const float beta = 0.5;

    auto ref_a = new float[ah * aw];
    auto ref_b = new float[aw * bw];
    auto ref_c = new float[ah * bw];
    for (int i = 0; i < ah * aw; i++)
        ref_a[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    for (int i = 0; i < aw * bw; i++)
        ref_b[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    for (int i = 0; i < ah * bw; i++)
        ref_c[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);

    float *a, *b, *c;
    hipMalloc(&a, ah * aw * sizeof(float));
    hipMalloc(&b, aw * bw * sizeof(float));
    hipMalloc(&c, ah * bw * sizeof(float));
    hipMemcpy(a, ref_a, ah * aw * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b, ref_b, aw * bw * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c, ref_c, ah * bw * sizeof(float), hipMemcpyHostToDevice);

    matmul(ref_a, ah, aw, ref_b, bw, ref_c, alpha, beta);
    auto timems = matmul_cu(a, ah, aw, b, bw, c, alpha, beta);
    float total_GBytes = (ah * aw + aw * bw + ah * bw + ah * bw) * sizeof(float) / 1024.0 / 1024 / 1024;
    std::cout << total_GBytes / (timems/1000.0) << " GBPS\n";
    std::cout << ah * aw * bw / (timems/1000.0) /1000000000000 << " TFLOPS\n";

    auto out_c = new float[ah * bw];
    hipMemcpy(out_c, c, ah * bw * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < ah * bw; i++) {
        auto diff = out_c[i] - ref_c[i];
        diff = diff > 0 ? diff : -diff;
        // std::cout<<diff<<"\n";
        if (diff > 0.1)
            return 1;
    }
    std::cout << "ok\n";

    hipFree(a);
    hipFree(b);
    hipFree(c);
    delete ref_a;
    delete ref_b;
    delete ref_c;
    delete out_c;
    return 0;
}
