#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <random>
#include <vector>
#include <assert.h>
#include <mma.h>
#include <hip/hip_fp16.h>
#include "utils.h"

template <typename scalar_t,
          int BLOCK_M_LANES, int BLOCK_N_LANES,
          int LANE_M_WARPS, int LANE_N_WARPS,
          int WARP_M_THREADS, int WARP_N_THREADS,
          int VEC_M, int VEC_N,
          int PAD = 8>
__global__ __launch_bounds__(256) void gemm_cuda_kernel(
    scalar_t *__restrict__ out,
    const scalar_t *__restrict__ a,
    const scalar_t *__restrict__ b,
    const int m, const int n, const int k,
    const scalar_t alpha,
    const scalar_t beta) {
    constexpr int BLOCK_K = 32;
    static_assert(LANE_M_WARPS * LANE_N_WARPS == 8);
    static_assert(WARP_M_THREADS * WARP_N_THREADS == 32);
    constexpr int WARP_M = WARP_M_THREADS * VEC_M;
    constexpr int WARP_N = WARP_N_THREADS * VEC_N;
    static_assert(WARP_M == 16);
    static_assert(WARP_N == 16);
    constexpr int LANE_M = LANE_M_WARPS * WARP_M;
    constexpr int LANE_N = LANE_N_WARPS * WARP_N;
    constexpr int BLOCK_M = BLOCK_M_LANES * LANE_M;
    constexpr int BLOCK_N = BLOCK_N_LANES * LANE_N;

    // idx
    auto tid = threadIdx.x;
    auto wid = tid >> 5;
    // auto w_tid = tid & 31;
    auto block_y = blockIdx.y;
    auto block_x = blockIdx.z * gridDim.x + blockIdx.x;

    // slm
    __shared__ scalar_t as[2][BLOCK_M * (BLOCK_K + PAD)];
    __shared__ scalar_t bs[2][BLOCK_K * (BLOCK_N + PAD)];

    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, scalar_t, nvcuda::wmma::row_major> a_frag[2][BLOCK_M_LANES];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, scalar_t, nvcuda::wmma::row_major> b_frag[2][BLOCK_N_LANES];
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> o_frag[BLOCK_M_LANES][BLOCK_N_LANES];
#pragma unroll
    for (int i = 0; i < BLOCK_M_LANES; i++) {
#pragma unroll
        for (int j = 0; j < BLOCK_N_LANES; j++) {
            nvcuda::wmma::fill_fragment(o_frag[i][j], 0.0);
        }
    }

    constexpr int LDG_VEC_SIZE = 8;
    using ldg_vec_t = aligned_array<scalar_t, LDG_VEC_SIZE>;
    constexpr int LDG_A_X_THREADS = BLOCK_K / LDG_VEC_SIZE;
    constexpr int LDG_B_X_THREADS = BLOCK_N / LDG_VEC_SIZE;
    auto ldg_a_vec_idx = tid % LDG_A_X_THREADS;
    auto ldg_b_vec_idx = tid % LDG_B_X_THREADS;
    constexpr int LDG_REG_A_COUNT = BLOCK_M * BLOCK_K / LDG_VEC_SIZE / 256;
    constexpr int LDG_REG_B_COUNT = BLOCK_K * BLOCK_N / LDG_VEC_SIZE / 256;
    static_assert(LDG_REG_A_COUNT >= 1 && LDG_REG_B_COUNT >= 1);

    int write_stage_idx = 0;
    int read_stage_idx = 1;

    for (
        int a_begin = block_y * BLOCK_M * k, b_begin = block_x * BLOCK_N;
        a_begin < block_y * BLOCK_M * k + k;
        a_begin += BLOCK_K, b_begin += BLOCK_K * n) {
        {
            ldg_vec_t ldg_a_reg[LDG_REG_A_COUNT];
            ldg_vec_t ldg_b_reg[LDG_REG_B_COUNT];
#pragma unroll
            for (int i = 0; i < LDG_REG_A_COUNT; i++) {
                auto idx = 256 * i + tid;
                ldg_a_reg[i] = reinterpret_cast<ldg_vec_t *>(const_cast<scalar_t *>(a) + a_begin + (idx / LDG_A_X_THREADS) * k)[ldg_a_vec_idx];
            }
#pragma unroll
            for (int i = 0; i < LDG_REG_B_COUNT; i++) {
                auto idx = 256 * i + tid;
                ldg_b_reg[i] = reinterpret_cast<ldg_vec_t *>(const_cast<scalar_t *>(b) + b_begin + (idx / LDG_B_X_THREADS) * n)[ldg_b_vec_idx];
            }
            auto as_vec = reinterpret_cast<ldg_vec_t *>(as[write_stage_idx]);
            auto bs_vec = reinterpret_cast<ldg_vec_t *>(bs[write_stage_idx]);
#pragma unroll
            for (int i = 0; i < LDG_REG_A_COUNT; i++) {
                int y = (256 * i + tid) / LDG_A_X_THREADS;
                as_vec[y * ((BLOCK_K + PAD) / LDG_VEC_SIZE) + ldg_a_vec_idx] = ldg_a_reg[i];
            }
#pragma unroll
            for (int i = 0; i < LDG_REG_B_COUNT; i++) {
                int y = (256 * i + tid) / LDG_B_X_THREADS;
                bs_vec[y * ((BLOCK_N + PAD) / LDG_VEC_SIZE) + ldg_b_vec_idx] = ldg_b_reg[i];
            }
            read_stage_idx ^= 1;
            write_stage_idx ^= 1;
            __syncthreads();
        }

        {
            auto a_ptr = as[read_stage_idx];
            auto b_ptr = bs[read_stage_idx];
            auto warp_y = wid / LANE_N_WARPS * WARP_M;
            auto warp_x = wid % LANE_N_WARPS * WARP_N;

#pragma unroll
            for (int i = 0; i < BLOCK_M_LANES; i++) {
                auto y = i * LANE_M + warp_y;
                nvcuda::wmma::load_matrix_sync(a_frag[0][i], a_ptr + y * (BLOCK_K + PAD), BLOCK_K + PAD);
                nvcuda::wmma::load_matrix_sync(a_frag[1][i], a_ptr + y * (BLOCK_K + PAD) + 16, BLOCK_K + PAD);
            }
#pragma unroll
            for (int j = 0; j < BLOCK_N_LANES; j++) {
                auto x = j * LANE_N + warp_x;
                nvcuda::wmma::load_matrix_sync(b_frag[0][j], b_ptr + x, BLOCK_N + PAD);
                nvcuda::wmma::load_matrix_sync(b_frag[1][j], b_ptr + x + 16 * (BLOCK_N + PAD), BLOCK_N + PAD);
            }
#pragma unroll
            for (int i = 0; i < BLOCK_M_LANES; i++) {
#pragma unroll
                for (int j = 0; j < BLOCK_N_LANES; j++) {
                    nvcuda::wmma::mma_sync(o_frag[i][j], a_frag[0][i], b_frag[0][j], o_frag[i][j]);
                    nvcuda::wmma::mma_sync(o_frag[i][j], a_frag[1][i], b_frag[1][j], o_frag[i][j]);
                }
            }
        }
    }

    { // write back
        auto out_warp_y = block_y * BLOCK_M + wid / LANE_N_WARPS * WARP_M;
        auto out_warp_x = block_x * BLOCK_N + wid % LANE_N_WARPS * WARP_N;
#pragma unroll
        for (int i = 0; i < BLOCK_M_LANES; i++) {
#pragma unroll
            for (int j = 0; j < BLOCK_N_LANES; j++) {
                auto y = out_warp_y + i * LANE_M;
                auto x = out_warp_x + j * LANE_N;
                if (y < m && x < n) {
                    auto out_offset = y * n + x;
                    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, scalar_t> c_frag;
                    nvcuda::wmma::load_matrix_sync(c_frag, out + out_offset, n, nvcuda::wmma::mem_row_major);
                    for (int k = 0; k < c_frag.num_elements; k++) {
                        c_frag.x[k] = alpha * (scalar_t)o_frag[i][j].x[k] + beta * c_frag.x[k];
                    }
                    nvcuda::wmma::store_matrix_sync(out + out_offset, c_frag, n, nvcuda::wmma::mem_row_major);
                    __syncthreads();
                }
            }
        }
    }
}

template <typename scalar_t, int BLOCK_M, int BLOCK_N>
float gemm_cuda_impl(
    scalar_t *out,
    const scalar_t *a,
    const scalar_t *b,
    const int m, const int n, const int k,
    const scalar_t alpha,
    const scalar_t beta) {
    assert(m % 16 == 0);
    assert(n % 16 == 0);
    assert(k % 32 == 0);
    int m_blocks = (m + BLOCK_M - 1) / BLOCK_M;
    int n_blocks = (n + BLOCK_N - 1) / BLOCK_N;
    constexpr int ZSPLIT = 32;
    int split_num = (n_blocks + ZSPLIT - 1) / ZSPLIT;
    dim3 block(256);
    dim3 grid((n_blocks + split_num - 1) / split_num, m_blocks, split_num);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    gemm_cuda_kernel<scalar_t, /*BLOCK_M_LANES*/ BLOCK_M / 32, /*BLOCK_N_LANES*/ BLOCK_N / 64, /*LANE_M_WARPS*/ 2, /*LANE_N_WARPS*/ 4,
                     /*WARP_M_THREADS*/ 8, /*WARP_N_THREADS*/ 4, /*VEC_M*/ 2, /*VEC_N*/ 4><<<grid, block>>>(out, a, b, m, n, k, alpha, beta);

    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    return milliseconds;
}

template <typename scalar_t>
float gemm_cuda(
    scalar_t *out,
    const scalar_t *a,
    const scalar_t *b,
    const int m, const int n, const int k,
    const scalar_t alpha,
    const scalar_t beta) {
    auto min_size = std::min(m, n);
    if (min_size <= 1024) {
        return gemm_cuda_impl<scalar_t, 64, 64>(out, a, b, m, n, k, alpha, beta);
    } else if (min_size <= 2048) {
        return gemm_cuda_impl<scalar_t, 64, 128>(out, a, b, m, n, k, alpha, beta);
    } else if (min_size <= 4096) {
        return gemm_cuda_impl<scalar_t, 64, 128>(out, a, b, m, n, k, alpha, beta);
    } else {
        return gemm_cuda_impl<scalar_t, 128, 128>(out, a, b, m, n, k, alpha, beta);
    }
}

template <typename scalar_t>
__global__ void gemm_cuda_ref_kernel(
    scalar_t *out,
    const scalar_t *a,
    const scalar_t *b,
    const int m, const int n, const int k,
    const scalar_t alpha,
    const scalar_t beta) {
    auto mi = blockIdx.y * 32 + threadIdx.y;
    auto ni = blockIdx.x * 32 + threadIdx.x;
    if (mi < m && ni < n) {
        float acc = 0.f;
        for (int ki = 0; ki < k; ki++) {
            acc += (float)a[mi * k + ki] * (float)b[ki * n + ni];
        }
        auto r = (float)alpha * acc;
        out[mi * n + ni] = r + (float)beta * (float)out[mi * n + ni];
    }
}

template <typename scalar_t = float>
void gemm_cuda_ref(
    scalar_t *out,
    const scalar_t *a,
    const scalar_t *b,
    const int m, const int n, const int k,
    const scalar_t alpha,
    const scalar_t beta) {
    dim3 block(32, 32);
    dim3 grid((n + 32 - 1) / 32, (m + 32 - 1) / 32);
    gemm_cuda_ref_kernel<scalar_t><<<grid, block>>>(out, a, b, m, n, k, alpha, beta);
    hipDeviceSynchronize();
}

struct gemm_sizes {
    int m, n, k;
    float alpha, beta;
    gemm_sizes(int m_, int n_, int k_, float a, float b) :
        m(m_), n(n_), k(k_), alpha(a), beta(b) {
    }
};

int main() {
    std::cout << "hgemm_wmma\n";
    using scalar_t = __half;

    std::vector<gemm_sizes> sizes;
    sizes.push_back(gemm_sizes(512, 512, 512, 0.5, 0.5));
    sizes.push_back(gemm_sizes(1024, 1024, 1024, 0.5, 0.5));
    sizes.push_back(gemm_sizes(1024 + 16, 1024 + 16, 64, 0.5, 0.5));
    sizes.push_back(gemm_sizes(2048, 2048, 2048, 0.5, 0.5));
    sizes.push_back(gemm_sizes(4096, 4096, 4096, 0.5, 0.5));
    sizes.push_back(gemm_sizes(8192, 8192, 8192, 0.5, 0.5));
    sizes.push_back(gemm_sizes(1 << 14, 1 << 14, 1 << 14, 0.5, 0.5));

    for (auto size : sizes) {
        int m = size.m;
        int n = size.n;
        int k = size.k;
        auto alpha = size.alpha;
        auto beta = size.beta;

        std::cout << "m=" << m << ", n=" << n << ", k=" << k
                  << ", alpha=" << alpha << ", beta=" << beta << "\n";

        auto a_cpu = new scalar_t[m * k];
        auto b_cpu = new scalar_t[k * n];
        auto out_cpu = new scalar_t[m * n];
        for (int i = 0; i < m * k; i++)
            a_cpu[i] = static_cast<scalar_t>((float)rand() / RAND_MAX);
        for (int i = 0; i < k * n; i++)
            b_cpu[i] = static_cast<scalar_t>((float)rand() / RAND_MAX);
        for (int i = 0; i < m * n; i++)
            out_cpu[i] = static_cast<scalar_t>((float)rand() / RAND_MAX);

        scalar_t *a_cuda, *b_cuda, *out_cuda;
        hipMalloc(&a_cuda, m * k * sizeof(scalar_t));
        hipMalloc(&b_cuda, k * n * sizeof(scalar_t));
        hipMalloc(&out_cuda, m * n * sizeof(scalar_t));
        hipMemcpy(a_cuda, a_cpu, m * k * sizeof(scalar_t), hipMemcpyHostToDevice);
        hipMemcpy(b_cuda, b_cpu, k * n * sizeof(scalar_t), hipMemcpyHostToDevice);
        hipMemcpy(out_cuda, out_cpu, m * n * sizeof(scalar_t), hipMemcpyHostToDevice);

        scalar_t *a_cuda_ref, *b_cuda_ref, *out_cuda_ref;
        hipMalloc(&a_cuda_ref, m * k * sizeof(scalar_t));
        hipMalloc(&b_cuda_ref, k * n * sizeof(scalar_t));
        hipMalloc(&out_cuda_ref, m * n * sizeof(scalar_t));
        hipMemcpy(a_cuda_ref, a_cpu, m * k * sizeof(scalar_t), hipMemcpyHostToDevice);
        hipMemcpy(b_cuda_ref, b_cpu, k * n * sizeof(scalar_t), hipMemcpyHostToDevice);
        hipMemcpy(out_cuda_ref, out_cpu, m * n * sizeof(scalar_t), hipMemcpyHostToDevice);

        gemm_cuda_ref<scalar_t>(out_cuda_ref, a_cuda_ref, b_cuda_ref, m, n, k, alpha, beta);
        auto timems = gemm_cuda<scalar_t>(out_cuda, a_cuda, b_cuda, m, n, k, alpha, beta);

        double total_gbytes = ((double)m * k + k * n + m * n + m * n) * sizeof(scalar_t) / 1000.0 / 1000 / 1000;
        std::cout << timems << " ms, " << total_gbytes / (timems / 1000.0) << " gbps, ";

        double tflops = ((double)2 * m * n * k) / (timems / 1000) * 1e-12;
        std::cout << tflops << " tflops\n";

        using MaxDiff = CompareMaxdiff<scalar_t>;
        auto diff = MaxDiff(out_cuda_ref, MaxDiff::CUDA, out_cuda, MaxDiff::CUDA, m * n);
        std::cout << "maxdiff: " << diff() << std::endl;

        hipFree(a_cuda);
        hipFree(b_cuda);
        hipFree(out_cuda);
        hipFree(a_cuda_ref);
        hipFree(b_cuda_ref);
        hipFree(out_cuda_ref);
        delete[] a_cpu;
        delete[] b_cpu;
        delete[] out_cpu;
    }
    return 0;
}
