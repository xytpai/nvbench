#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <random>
using namespace std;

void matmul(const float *a, int ah, int aw, const float *b, int bw, float *c, float alpha, float beta) {
    for (int i = 0; i < ah; i++) {
        for (int j = 0; j < bw; j++) {
            float sum = 0;
            for (int k = 0; k < aw; k++)
                sum += a[i * aw + k] * b[k * bw + j];
            c[i * bw + j] = alpha * sum + beta * c[i * bw + j];
        }
    }
}

template<int BM=128, int BN=128, int BK=16, int TM=4, int TN=4, int SQRT_BSIZE=16>
__global__ void matmul_kernel(
    const float *A, int Ah, int Aw,
    const float *B, int Bw,
    float *C,
    float alpha, float beta) {
    auto lid = threadIdx.x;

    auto block_y_begin = (blockIdx.y * BM)*Aw;
    auto block_y_end = block_y_begin + Aw;
    auto block_y_step = BK;
    auto block_x_begin = blockIdx.x * BN;
    auto block_x_step = BK * Bw;

    constexpr int LDG_REG_A_COUNT = BM * BK / 4 / SQRT_BSIZE / SQRT_BSIZE;
    constexpr int LDG_REG_B_COUNT = BN * BK / 4 / SQRT_BSIZE / SQRT_BSIZE;
    float4 ldg_a_reg[LDG_REG_A_COUNT];
    float4 ldg_b_reg[LDG_REG_B_COUNT];
    constexpr int LDG_A_X_CT = BK / 4;
    constexpr int LDG_B_X_CT = BN / 4;
    auto lid_mod_LDG_A_X_CT = lid % LDG_A_X_CT;
    auto lid_mod_LDG_B_X_CT = lid % LDG_B_X_CT;

    float4 a_reg[LDG_REG_A_COUNT][LDG_REG_B_COUNT];
    float4 b_reg[LDG_REG_A_COUNT][LDG_REG_B_COUNT];

    float tmp[TM * TN] = {0.0};

    int write_stage_idx = 1; //ping pong switch
    int read_stage_idx = 0;
    for (int a_begin = block_y_begin, b_begin = block_x_begin; 
        a_begin < block_y_end; a_begin += block_y_step, b_begin += block_x_step) 
    {
        __shared__ float4 As[2][BK * BM / 4];
        __shared__ float4 Bs[2][BK * BN / 4];
#pragma unroll
        for(int i=0; i<LDG_REG_A_COUNT; i++) {
            ldg_a_reg[i] = reinterpret_cast<float4*>(const_cast<float*>(A) + a_begin + ((blockDim.x * i + lid) / LDG_A_X_CT) * Aw)[lid_mod_LDG_A_X_CT];
        }
#pragma unroll
        for(int i=0; i<LDG_REG_B_COUNT; i++) {
            ldg_b_reg[i] = reinterpret_cast<float4*>(const_cast<float*>(B) + b_begin + ((blockDim.x * i + lid) / LDG_B_X_CT) * Bw)[lid_mod_LDG_B_X_CT];
        }
#pragma unroll
        for(int i=0; i<LDG_REG_A_COUNT; i++) {
            auto y = (blockDim.x * i + lid) / LDG_A_X_CT;
#pragma unroll
            for(int j=0; j<4; j++) {
                reinterpret_cast<float*>(&As[read_stage_idx][(lid_mod_LDG_A_X_CT * 4 + j) * (BM/4) + y/4].x)[y%4] = reinterpret_cast<float*>(&ldg_a_reg[i].x)[j];
            }
        }
#pragma unroll
        for(int i=0; i<LDG_REG_B_COUNT; i++) {
            Bs[read_stage_idx][blockDim.x * i + lid] = ldg_b_reg[i];
        }
        __syncthreads();

#pragma unroll
        for(int k=0; k<BK; k++) {
#pragma unroll
            for(int ia=0; ia<LDG_REG_A_COUNT; ia++) {
#pragma unroll
                for(int ib=0; ib<LDG_REG_B_COUNT; ib++) {
                    a_reg[ia][ib] = As[read_stage_idx][k * BM + ia * ];
                }
            }
        }
        // if (lid == 0) {
        // auto As_ = reinterpret_cast<float*>(As);
        // auto Bs_ = reinterpret_cast<float*>(Bs);
        // for(int k=0; k<BK; k++) {
        //     for(int m=0; m<BM; m++) {
        //         for(int n=0; n<BN; n++) {
        //             C[(blockIdx.y * BM + m) * Bw + (blockIdx.x * BN + n)] += As_[k * BM + m] * Bs_[k * BN + n];
        //         }
        //     }
        // }
        // }

        __syncthreads();
   
    }
}

float matmul_cu(const float *a, int ah, int aw, const float *b, int bw, float *c, float alpha, float beta) {
    dim3 threadsPerBlock(16 * 16);
    dim3 numBlocks(ah / 128, bw / 128);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matmul_kernel<<<numBlocks, threadsPerBlock>>>(a, ah, aw, b, bw, c, alpha, beta);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << milliseconds << " ms" << std::endl;
    return milliseconds;
}

int main() {
    const int ah = 1024;
    const int aw = 1024;
    const int bw = 1024;
    const float alpha = 1.0;
    const float beta = 0.0;

    auto ref_a = new float[ah * aw];
    auto ref_b = new float[aw * bw];
    auto ref_c = new float[ah * bw];
    for (int i = 0; i < ah * aw; i++)
        ref_a[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    for (int i = 0; i < aw * bw; i++)
        ref_b[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    for (int i = 0; i < ah * bw; i++)
        ref_c[i] = 0; //static_cast<float>(rand()) / static_cast<float>(RAND_MAX);

    float *a, *b, *c;
    hipMalloc(&a, ah * aw * sizeof(float));
    hipMalloc(&b, aw * bw * sizeof(float));
    hipMalloc(&c, ah * bw * sizeof(float));
    hipMemcpy(a, ref_a, ah * aw * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b, ref_b, aw * bw * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c, ref_c, ah * bw * sizeof(float), hipMemcpyHostToDevice);

    matmul(ref_a, ah, aw, ref_b, bw, ref_c, alpha, beta);
    auto timems = matmul_cu(a, ah, aw, b, bw, c, alpha, beta);
    float total_GBytes = (ah * aw + aw * bw + ah * bw + ah * bw) * sizeof(float) / 1024.0 / 1024 / 1024;
    std::cout << total_GBytes / (timems/1000.0) << " gbps\n";

    auto out_c = new float[ah * bw];
    hipMemcpy(out_c, c, ah * bw * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < ah * bw; i++) {
        auto diff = out_c[i] - ref_c[i];
        diff = diff > 0 ? diff : -diff;
        // std::cout<<diff<<"\n";
        if (diff > 0.1)
            return 1;
    }
    std::cout << "ok\n";

    hipFree(a);
    hipFree(b);
    hipFree(c);
    delete ref_a;
    delete ref_b;
    delete ref_c;
    delete out_c;
    return 0;
}
