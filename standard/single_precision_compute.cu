#include <hip/hip_runtime.h>

#include <iostream>
#include <random>
using namespace std;

template <int LOOP>
__global__ void fmad_loop_kernel(float *x) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    float a = x[index], b = -1.0f;
    for (int i = 0; i < LOOP; i++) {
        for (int j = 0; j < LOOP; j++) {
            a = a * b + b;
        }
    }
    x[index] = a;
}

template <int LOOP, int block_size, int num_blocks>
float fmad_test() {
    dim3 threadsPerBlock(block_size);
    dim3 numBlocks(num_blocks);
    constexpr int n = block_size * num_blocks;
    auto x = new float[n];
    float *dx;
    hipMalloc(&dx, n * sizeof(float));
    hipMemcpy(dx, x, n * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    fmad_loop_kernel<LOOP><<<numBlocks, threadsPerBlock>>>(dx);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(x, dx, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dx);
    delete[] x;
    return ms;
}

int main() {
    constexpr int LOOP = 10000;
    constexpr int block_size = 256;
    constexpr int num_blocks = 2048;
    for (int i = 0; i < 3; i++) {
        auto timems = fmad_test<LOOP, block_size, num_blocks>();
        auto tflops =
            2.0 * LOOP * LOOP * num_blocks * block_size / (timems / 1000) * 1e-12;
        auto arithmetic_intensity = 2.0f * LOOP * LOOP / (sizeof(float) * 2);
        std::cout << "arithmetic_intensity: " << arithmetic_intensity << " FLOP/Byte.  |  COMPUTE:";
        std::cout << tflops << " TFLOPS" << std::endl;
    }
    return 0;
}
