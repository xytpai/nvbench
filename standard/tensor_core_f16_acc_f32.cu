#include <hip/hip_runtime.h>

#include <iostream>
#include <random>
#include <mma.h>
#include <hip/hip_fp16.h>
using namespace std;

template <typename scalar_t, int LOOP>
__global__ void wmma_loop_kernel(scalar_t *input, float *output, int stride_in_elem) {
    // scalar_t: __half or __bfloat16
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int wid = index / 32;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, scalar_t, nvcuda::wmma::row_major> a_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, scalar_t, nvcuda::wmma::row_major> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> o_frag;
    nvcuda::wmma::fill_fragment(o_frag, 0.0);
    nvcuda::wmma::load_matrix_sync(a_frag, input + wid * 16, stride_in_elem);
    nvcuda::wmma::load_matrix_sync(b_frag, input + wid * 16, stride_in_elem);
    for (int i = 0; i < LOOP; i++) {
        nvcuda::wmma::mma_sync(o_frag, a_frag, b_frag, o_frag);
    }
    nvcuda::wmma::store_matrix_sync(output + wid * 16, o_frag, stride_in_elem, nvcuda::wmma::mem_row_major);
}

template <int LOOP, int num_blocks>
float wmma_test() {
    dim3 threadsPerBlock(256);
    dim3 numBlocks(num_blocks);
    constexpr int n = 16 * 256 * num_blocks;
    auto input = new __half[n];
    auto output = new float[n];
    __half *dinput;
    float *doutput;
    hipMalloc(&dinput, n * sizeof(__half));
    hipMemcpy(dinput, input, n * sizeof(__half), hipMemcpyHostToDevice);
    hipMalloc(&doutput, n * sizeof(float));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    wmma_loop_kernel<__half, LOOP><<<numBlocks, threadsPerBlock>>>(dinput, doutput, 256 * num_blocks);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(output, doutput, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dinput);
    hipFree(doutput);
    delete[] input;
    delete[] output;
    return ms;
}

int main() {
    constexpr int LOOP = 1000000;
    constexpr int num_blocks = 4096;
    constexpr int warps_per_block = 256 / 32;
    for (int i = 0; i < 3; i++) {
        auto timems = wmma_test<LOOP, num_blocks>();
        auto tflops =
            ((double)2 * 16 * 16 * 16) * LOOP * num_blocks * warps_per_block / (timems / 1000) * 1e-12;
        std::cout << tflops << " TFLOPS" << std::endl;
    }
    return 0;
}
